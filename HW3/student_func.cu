#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__global__ void reduce_minimum(float * d_out, const float * const d_in, const size_t numItem) {
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid  = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = 99999999999.0f;
  if (myId < numItem)
    sdata[tid] = d_in[myId];

  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
        sdata[tid] = min(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0) {
    d_out[blockIdx.x] = sdata[0];
  }
}

__global__ void reduce_maximum(float * d_out, const float * const d_in, const size_t numItem) {
  // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid  = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = -99999999999.0f;
  if (myId < numItem)
    sdata[tid] = d_in[myId];

  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] = max(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0) {
    d_out[blockIdx.x] = sdata[0];
  }
}

__global__ void histogram(unsigned int *d_bins, const float * const d_in, const size_t numBins, const float min_logLum, const float range, const size_t numRows, const size_t numCols) {
  
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  if (myId >= (numRows * numCols))
    return;

  float myItem = d_in[myId];
  int myBin = (myItem - min_logLum) / range * numBins;
  atomicAdd(&(d_bins[myBin]), 1);
}

__global__ void scan(unsigned int *d_out, unsigned int *d_sums, const unsigned int * const d_in, const unsigned int numBins, const unsigned int numElems)  {

  extern __shared__ float sdata[];
  int myId = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;
  int offset = 1;

  // load two items per thread into shared memory
  if ((2 * myId) < numBins) {
    sdata[2 * tid] = d_in[2 * myId];
  }
  else {
    sdata[2 * tid] = 0;
  }
  
  if ((2 * myId + 1) < numBins) {
    sdata[2 * tid + 1] = d_in[2 * myId + 1];
  }
  else {
    sdata[2 * tid + 1] = 0;
  }

 	// Reduce
  for (unsigned int d = numElems >> 1; d > 0; d >>= 1) {
    if (tid < d)  {
      int ai = offset * (2 * tid + 1) - 1;
      int bi = offset * (2 * tid + 2) - 1;
      sdata[bi] += sdata[ai];
    }
    offset *= 2;
    __syncthreads();
  }
    
  // clear the last element
  if (tid == 0) {
    d_sums[blockIdx.x] = sdata[numElems - 1];
    sdata[numElems - 1] = 0;
  }
  
  // Down Sweep
  for (unsigned int d = 1; d < numElems; d *= 2) {
    offset >>= 1;
    if (tid < d) {
      int ai = offset * (2 * tid + 1) - 1;
      int bi = offset * (2 * tid + 2) - 1;
 	    float t = sdata[ai];
      sdata[ai] = sdata[bi];
      sdata[bi] += t;
    }
    __syncthreads();
  }
 
  // write the output to global memory
  if ((2 * myId) < numBins) {
    d_out[2 * myId] = sdata[2 * tid];
  }
  if ((2 * myId + 1) < numBins) {
    d_out[2 * myId + 1] = sdata[2 * tid + 1];
  }
}

// This version only works for one single block! The size of the array of items
__global__ void scan2(unsigned int *d_out, const unsigned int * const d_in, const unsigned int numBins, const unsigned int numElems)  {

  extern __shared__ float sdata[];
  int tid = threadIdx.x;
  int offset = 1;

  // load two items per thread into shared memory
  if ((2 * tid) < numBins) {
    sdata[2 * tid] = d_in[2 * tid];  
  }
  else {
    sdata[2 * tid] = 0;
  }

  if ((2 * tid + 1) < numBins) {
    sdata[2 * tid + 1] = d_in[2 * tid + 1];  
  }
  else {
    sdata[2 * tid + 1] = 0;
  }

 	// Reduce
  for (unsigned int d = numElems >> 1; d > 0; d >>= 1) {
    if (tid < d)  {
      int ai = offset * (2 * tid + 1) - 1;
      int bi = offset * (2 * tid + 2) - 1;
      sdata[bi] += sdata[ai];
    }
    offset *= 2;
    __syncthreads();
  }
    
  // clear the last element
  if (tid == 0) {
    sdata[numElems - 1] = 0;
  }
  
  // Down Sweep
  for (unsigned int d = 1; d < numElems; d *= 2) {
    offset >>= 1;
    if (tid < d) {
      int ai = offset * (2 * tid + 1) - 1;
      int bi = offset * (2 * tid + 2) - 1;
 	    float t = sdata[ai];
      sdata[ai] = sdata[bi];
      sdata[bi] += t;
    }
    __syncthreads();
  }
 
  // write the output to global memory
  if ((2 * tid) < numBins) {
    d_out[2 * tid] = sdata[2 * tid];
  }

  if ((2 * tid + 1) < numBins) {
    d_out[2 * tid + 1] = sdata[2 * tid + 1];
  }
}

__global__ void add_scan(unsigned int *d_out, const unsigned int * const d_in, const unsigned int numBins) {

  if (blockIdx.x == 0)
    return;

  int myId = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int myOffset = d_in[blockIdx.x];

  if ((2 * myId) < numBins) {
    d_out[2 * myId] += myOffset;
  }
  if ((2 * myId + 1) < numBins) {
    d_out[2 * myId + 1] += myOffset;
  }

}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  // Initialization
  unsigned int numItem = numRows * numCols;
  dim3 blockSize(256, 1, 1);
  dim3 gridSize(numItem / blockSize.x + 1, 1, 1);
    
  float * d_inter_min;
  float * d_inter_max;
  unsigned int * d_histogram;
  unsigned int * d_sums;
  unsigned int * d_incr;

  checkCudaErrors(hipMalloc(&d_inter_min, sizeof(float) * gridSize.x));
  checkCudaErrors(hipMalloc(&d_inter_max, sizeof(float) * gridSize.x));
  checkCudaErrors(hipMalloc(&d_histogram, sizeof(unsigned int) * numBins));
  checkCudaErrors(hipMemset(d_histogram, 0, sizeof(unsigned int) * numBins));
     
  // Step 1: Reduce (min and max). It could be done in one step only!
  reduce_minimum<<<gridSize, blockSize, sizeof(float) * blockSize.x>>>(d_inter_min, d_logLuminance, numItem);
  reduce_maximum<<<gridSize, blockSize, sizeof(float) * blockSize.x>>>(d_inter_max, d_logLuminance, numItem);
  numItem = gridSize.x;
  gridSize.x = numItem / blockSize.x + 1;

  while (numItem > 1) {
    reduce_minimum<<<gridSize, blockSize, sizeof(float) * blockSize.x>>>(d_inter_min, d_inter_min, numItem);
    reduce_maximum<<<gridSize, blockSize, sizeof(float) * blockSize.x>>>(d_inter_max, d_inter_max, numItem);
    numItem = gridSize.x;
    gridSize.x = numItem / blockSize.x + 1;
  }

  // Step 2: Range
  checkCudaErrors(hipMemcpy(&min_logLum, d_inter_min, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_inter_max, sizeof(float), hipMemcpyDeviceToHost));

  float range = max_logLum - min_logLum;

  // Step 3: Histogram
  gridSize.x = numRows * numCols / blockSize.x + 1;
  histogram<<<gridSize, blockSize>>>(d_histogram, d_logLuminance, numBins, min_logLum, range, numRows, numCols);

  // Step 4: Exclusive scan - Blelloch
  unsigned int numElems = 256;
  blockSize.x = numElems / 2;
  gridSize.x = numBins / numElems;
  if (numBins % numElems != 0)
    gridSize.x++;
  checkCudaErrors(hipMalloc(&d_sums, sizeof(unsigned int) * gridSize.x));
  checkCudaErrors(hipMemset(d_sums, 0, sizeof(unsigned int) * gridSize.x));

  // First-level scan to obtain the scanned blocks
  scan<<<gridSize, blockSize, sizeof(float) * numElems>>>(d_cdf, d_sums, d_histogram, numBins, numElems);

  // Second-level scan to obtain the scanned blocks sums
  numElems = gridSize.x;

  // Look for the next power of 2 (32 bits)
  unsigned int nextPow = numElems;
  nextPow--;
  nextPow = (nextPow >> 1) | nextPow;
  nextPow = (nextPow >> 2) | nextPow;
  nextPow = (nextPow >> 4) | nextPow;
  nextPow = (nextPow >> 8) | nextPow;
  nextPow = (nextPow >> 16) | nextPow;
  nextPow++;

  blockSize.x = nextPow / 2;
  gridSize.x = 1;
  checkCudaErrors(hipMalloc(&d_incr, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemset(d_incr, 0, sizeof(unsigned int) * numElems));
  scan2<<<gridSize, blockSize, sizeof(float) * nextPow>>>(d_incr, d_sums, numElems, nextPow);

  // Add scanned block sum i to all values of scanned block i
  numElems = 256;
  blockSize.x = numElems / 2;
  gridSize.x = numBins / numElems;
  if (numBins % numElems != 0)
    gridSize.x++;
  add_scan<<<gridSize, blockSize>>>(d_cdf, d_incr, numBins);

  // Clean memory
  checkCudaErrors(hipFree(d_inter_min));
  checkCudaErrors(hipFree(d_inter_max));
  checkCudaErrors(hipFree(d_histogram));
  checkCudaErrors(hipFree(d_sums));
  checkCudaErrors(hipFree(d_incr));
}
